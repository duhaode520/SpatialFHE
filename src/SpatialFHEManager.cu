#include "hip/hip_runtime.h"
//
// Created by ubuntu on 10/14/24.
//

#include "SpatialFHEManager.h"

SpatialFHE::CipherMat SpatialFHE::SpatialFHEManager::parallelOp2(
    CipherMat const &a,
    CipherMat const &b,
    std::function<CipherText(CipherText const &, CipherText const &)> func) const {
    validateSameSize(a, b);
    int size = a.getData().size();
    std::vector<CipherText> result(size);
    int numThreads = std::min(size, max_thread_num);

    if (numThreads == 1) {
        for (int i = 0; i < size; i++) {
            result[i] = func(a.getData()[i], b.getData()[i]);
        }
        return {a.getWidth(), a.getHeight(), result};
    }

    std::vector<std::thread> threads(numThreads);
    int chunkSize = size / numThreads;
    for (int i = 0; i < numThreads; i++) {
        threads[i] = std::thread([&, i]() {
            int start = i * chunkSize;
            int end = (i == numThreads - 1) ? size : start + chunkSize;
            for (int j = start; j < end; j++) {
                result[j] = func(a.getData()[j], b.getData()[j]);
            }
        });
    }
    for (auto &t : threads) {
        if (t.joinable())
            t.join();
    }
    return {a.getWidth(), a.getHeight(), result};
}

SpatialFHE::CipherMat SpatialFHE::SpatialFHEManager::parallelOp2(
    CipherMat const &a,
    std::vector<PlainText> const &plainVector,
    std::function<CipherText(CipherText const &, PlainText const &)> func) const {
    if (a.getData().size() != plainVector.size()) {
        throw std::invalid_argument("CipherMat CipherText vector size not match");
    }
    int size = a.getData().size();
    std::vector<CipherText> result(size);
    int numThreads = std::min(size, max_thread_num);

    if (numThreads == 1) {
        for (int i = 0; i < size; i++) {
            result[i] = func(a.getData()[i], plainVector[i]);
        }
        return {a.getWidth(), a.getHeight(), result};
    }

    std::vector<std::thread> threads(numThreads);
    int chunkSize = size / numThreads;
    for (int i = 0; i < numThreads; i++) {
        threads[i] = std::thread([&, i]() {
            int start = i * chunkSize;
            int end = (i == numThreads - 1) ? size : start + chunkSize;
            for (int j = start; j < end; j++) {
                result[j] = func(a.getData()[j], plainVector[j]);
            }
        });
    }
    for (auto &t : threads) {
        if (t.joinable())
            t.join();
    }
    return {a.getWidth(), a.getHeight(), result};
}

SpatialFHE::SpatialFHEManager::SpatialFHEManager() {
    max_thread_num = 8;
}

SpatialFHE::SpatialFHEManager::SpatialFHEManager(
    const std::string &publicKeyPath,
    const std::string &secretKeyPath,
    const std::string &paramsString,
    const std::string &serverURL,
    const HECrypto::HELibrary heLibrary,
    bool isInit,
    int max_thread_num = 8) :
        BaseFHEManager(publicKeyPath, secretKeyPath, paramsString, heLibrary, isInit),
        VectorFHEManager(publicKeyPath, secretKeyPath, serverURL, isInit) {
    this->max_thread_num = max_thread_num;
}

SpatialFHE::SpatialFHEManager::~SpatialFHEManager() = default;

SpatialFHE::CipherMat SpatialFHE::SpatialFHEManager::encryptMat(
    int width,
    int height,
    const std::vector<double> &data) {
    std::vector<PlainText> plain_vec = crypto->EncodeMany(data);
    std::vector<CipherText> cipher_vec;
    if (heLib == HECrypto::HELibrary::Phantom) {
        for (auto &plain : plain_vec) {
            cipher_vec.emplace_back(crypto->Encrypt(plain));
        }
    } else {
        cipher_vec.resize(plain_vec.size());
        int size = plain_vec.size();
        int numThreads = std::min(size, (int)std::thread::hardware_concurrency());

        std::vector<std::thread> threads(numThreads);
        int chunkSize = size / numThreads;

        for (int i = 0; i < numThreads; i++) {
            threads[i] = std::thread([&, i]() {
                int start = i * chunkSize;
                int end = (i == numThreads - 1) ? size : start + chunkSize;
                for (int j = start; j < end; j++) {
                    cipher_vec[j] = crypto->Encrypt(plain_vec[j]);
                }
            });
        }

        for (auto &t : threads) {
            if (t.joinable())
                t.join();
        }
    }

    return {width, height, cipher_vec};
}

std::vector<double> SpatialFHE::SpatialFHEManager::decryptMat(CipherMat &cipher_mat) {
    int width = cipher_mat.getWidth();
    int height = cipher_mat.getHeight();
    int size = cipher_mat.getData().size();

    std::vector<PlainText> plain_vec;
    if (heLib == HECrypto::HELibrary::Phantom) {
        for (auto &cipher : cipher_mat.getData()) {
            plain_vec.emplace_back(crypto->Decrypt(cipher));
        }
    } else {
        plain_vec.resize(size);
        int numThreads = std::min(size, (int)std::thread::hardware_concurrency());
        std::vector<std::thread> threads(numThreads);
        int chunkSize = size / numThreads;

        // 并行解密
        for (int i = 0; i < numThreads; i++) {
            threads[i] = std::thread([&, i]() {
                int start = i * chunkSize;
                int end = (i == numThreads - 1) ? size : start + chunkSize;
                for (int j = start; j < end; j++) {
                    plain_vec[j] = crypto->Decrypt(cipher_mat.getData()[j]);
                }
            });
        }

        for (auto &t : threads) {
            if (t.joinable())
                t.join();
        }
    }

    // 解码后的结果
    std::vector<double> result;
    for (auto &plain : plain_vec) {
        std::vector<double> tmp_vector;
        crypto->Decode(tmp_vector, plain);
        result.insert(result.end(), tmp_vector.begin(), tmp_vector.end());
    }

    // cut the result to width * height
    return std::vector<double>(result.begin(), result.begin() + width * height);
}

SpatialFHE::CipherMat SpatialFHE::SpatialFHEManager::addMat(CipherMat const &a, CipherMat const &b) {
    return parallelOp2(a, b, [this](CipherText const &x, CipherText const &y) { return crypto->Add(x, y); });
}

SpatialFHE::CipherMat SpatialFHE::SpatialFHEManager::addMatPlain(CipherMat const &a, const std::vector<double> &plain) {
    std::vector<PlainText> plain_vec = crypto->EncodeMany(plain);
    return parallelOp2(
        a, plain_vec, [this](CipherText const &x, PlainText const &y) { return crypto->AddPlain(x, y); });
}

SpatialFHE::CipherMat SpatialFHE::SpatialFHEManager::subMat(CipherMat const &a, CipherMat const &b) {
    return parallelOp2(a, b, [this](CipherText const &x, CipherText const &y) { return crypto->Subtract(x, y); });
}

SpatialFHE::CipherMat SpatialFHE::SpatialFHEManager::multiplyMat(CipherMat const &a, CipherMat const &b) {
    return parallelOp2(a, b, [this](CipherText const &x, CipherText const &y) { return crypto->Multiply(x, y); });
}

SpatialFHE::CipherMat SpatialFHE::SpatialFHEManager::multiplyMatPlain(
    CipherMat const &a,
    const std::vector<double> &plain) {
    std::vector<PlainText> plain_vec = crypto->EncodeMany(plain);
    return parallelOp2(
        a, plain_vec, [this](CipherText const &x, PlainText const &y) { return crypto->MultiplyPlain(x, y); });
}

void SpatialFHE::SpatialFHEManager::validateSameSize(SpatialFHE::CipherMat const &a, SpatialFHE::CipherMat const &b) {
    if (a.getWidth() != b.getWidth() || a.getHeight() != b.getHeight()) {
        throw std::invalid_argument("CipherMat size not match");
    }
    if (a.getData().size() != b.getData().size()) {
        throw std::invalid_argument("CipherMat CipherText vector size not match");
    }
}
