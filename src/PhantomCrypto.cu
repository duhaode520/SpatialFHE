#include "hip/hip_runtime.h"
#include "PhantomCrypto.cuh"
#include "FSManager.h"
#include "phantom.h"
#include <b64/decode.h>

using namespace std;
namespace SpatialFHE {
    void PhantomCrypto::update_encryption_params(CryptoParams &crypto_params) {
        this->params = make_shared<PhantomEncryptionParams>();
        if (crypto_params.find("PlaintextModulus") != crypto_params.end()) {
            this->params->plainModulus = crypto_params["PlaintextModulus"].GetInt64();
        }
        if (crypto_params.find("PolyModulusDegree") != crypto_params.end()) {
            this->params->polyModulusDegree = crypto_params["PolyModulusDegree"].GetInt64();
        }
        if (crypto_params.find("ScaleFactor") != crypto_params.end()) {
            this->params->scaleFactor = crypto_params["ScaleFactor"].GetInt64();
            this->params->scale = pow(2.0, this->params->scaleFactor);
        }
        if (crypto_params.find("SchemeType") != crypto_params.end()) {
            this->params->schemeType = parse_HE_scheme(crypto_params["SchemeType"].GetString());
        }
        if (crypto_params.find("CoeffModulusBits") != crypto_params.end()) {
            vector<long> lvec = to_long_vec(crypto_params["CoeffModulusBits"]);
            this->params->coeffModulusBits = vector<int>(lvec.begin(), lvec.end());
        }
        if (crypto_params.find("CoeffModulusPrimes") != crypto_params.end()) {
            vector<long> lvec = to_long_vec(crypto_params["CoeffModulusPrimes"]);
            this->params->coeffModulusPrimes = vector<ulong>(lvec.begin(), lvec.end());
        }
    }

    void PhantomCrypto::initPhantomParams() {
        phantom::scheme_type type = set_phantom_scheme(this->params->schemeType);
        phantom::EncryptionParameters encParams(type);
        encParams.set_poly_modulus_degree(this->params->polyModulusDegree);
        if (this->params->schemeType == HECrypto::HEScheme::BFV) {
            encParams.set_coeff_modulus(phantom::arith::CoeffModulus::BFVDefault(this->params->polyModulusDegree));
        } else if (this->params->schemeType == HECrypto::HEScheme::CKKS) {
            encParams.set_coeff_modulus(
                phantom::arith::CoeffModulus::Create(this->params->polyModulusDegree, this->params->coeffModulusBits));
            // TODO: check if batching is properly set
            this->batching = true;
        }

        vector<ulong> coeffmod_primes;
        vector<ulong> coeffmod_bits;
        for (auto &mod : encParams.coeff_modulus()) {
            coeffmod_primes.push_back(mod.value());
            coeffmod_bits.push_back(mod.bit_count());
        }

        this->params->coeffModulusBits = vector<int>(coeffmod_bits.begin(), coeffmod_bits.end());
        this->params->coeffModulusPrimes = coeffmod_primes;

        this->phantomParams = make_shared<phantom::EncryptionParameters>(encParams);
    }
    void PhantomCrypto::set_encoder(HEScheme scheme) {
        if (scheme == HEScheme::BFV) {
            this->batchEncoder = make_shared<PhantomBatchEncoder>(*this->context);
            this->slot_count = this->batchEncoder->slot_count();
        } else if (scheme == HEScheme::CKKS) {
            this->ckksEncoder = make_shared<PhantomCKKSEncoder>(*this->context);
            this->slot_count = this->ckksEncoder->slot_count();
        }
    }

    phantom::scheme_type PhantomCrypto::set_phantom_scheme(HEScheme scheme) {
        if (scheme == HECrypto::HEScheme::BFV) {
            return phantom::scheme_type::bfv;
        } else if (scheme == HECrypto::HEScheme::CKKS) {
            return phantom::scheme_type::ckks;
        } else {
            cerr << "Invalid scheme type" << endl;
            throw std::runtime_error("Invalid scheme type");
        }
    }
    void PhantomCrypto::parms_unify(PhantomCiphertext &src, PhantomCiphertext &dst) {
        while (src.chain_index() > dst.chain_index()) {
            phantom::rescale_to_next_inplace(*context, dst);
        }
        while (src.chain_index() < dst.chain_index()) {
            phantom::rescale_to_next_inplace(*context, src);
        }
    }

    void PhantomCrypto::parms_unify(PhantomPlaintext &src, PhantomCiphertext &dst) const {
        while (src.chain_index() < dst.chain_index()) {
            phantom::mod_switch_to_next_inplace(*context, src);
        }
    }

    void PhantomCrypto::_encrypt(PhantomCiphertext &ct, PhantomPlaintext const &pt) {
        publicKey.encrypt_asymmetric(*context, pt, ct);
    }

    void PhantomCrypto::_decrypt(PhantomPlaintext &pt, PhantomCiphertext const &ct) {
        secretKey.decrypt(*context, ct, pt);
    }

    void PhantomCrypto::_encode(PhantomPlaintext &pt, double d) {
        vector<double> vec(this->slot_count, d);
        this->ckksEncoder->encode(*context, vec, this->params->scale, pt);
    }

    void PhantomCrypto::_encode(PhantomPlaintext &pt, long l) {
        vector<ulong> vec(this->slot_count, static_cast<ulong>(l));
        this->batchEncoder->encode(*context, vec, pt);
    }

    void PhantomCrypto::_add(PhantomCiphertext &ct_1, PhantomCiphertext const &ct_2) {
        phantom::add_inplace(*context, ct_1, ct_2);
    }

    void PhantomCrypto::_add(PhantomCiphertext &result, PhantomCiphertext const &ct_1, PhantomCiphertext const &ct_2) {
        result = phantom::add(*context, ct_1, ct_2);
    }

    void PhantomCrypto::_add_plain(PhantomCiphertext &ct, PhantomPlaintext const &pt) {
        phantom::add_plain_inplace(*context, ct, pt);
    }

    void PhantomCrypto::_full_adder(
        std::vector<PhantomCiphertext> &result,
        std::vector<PhantomCiphertext> const &vec_ct_1,
        std::vector<PhantomCiphertext> const &vec_ct_2,
        size_t max_count) {
        throw std::logic_error("Not implemented");
    }

    void PhantomCrypto::_sub(PhantomCiphertext &ct_1, PhantomCiphertext const &ct_2) {
        phantom::sub_inplace(*context, ct_1, ct_2);
    }

    void PhantomCrypto::_sub(PhantomCiphertext &result, PhantomCiphertext const &ct_1, PhantomCiphertext const &ct_2) {
        result = phantom::sub(*context, ct_1, ct_2);
    }

    void PhantomCrypto::_multiply(PhantomCiphertext &ct_1, PhantomCiphertext const &ct_2) {
        phantom::multiply_and_relin_inplace(*context, ct_1, ct_2, relinKey);
        if (this->params->schemeType == HECrypto::HEScheme::CKKS) {
            phantom::rescale_to_next_inplace(*context, ct_1);
        }
    }

    void PhantomCrypto::_multiply(
        PhantomCiphertext &result,
        PhantomCiphertext const &ct_1,
        PhantomCiphertext const &ct_2) {
        result = phantom::multiply_and_relin(*context, ct_1, ct_2, relinKey);
        if (this->params->schemeType == HECrypto::HEScheme::CKKS) {
            phantom::rescale_to_next_inplace(*context, result);
        }
    }

    void PhantomCrypto::_multiply_plain(PhantomCiphertext &ct, PhantomPlaintext const &pt) {
        phantom::multiply_plain_inplace(*context, ct, pt);
        if (this->params->schemeType == HECrypto::HEScheme::CKKS) {
            phantom::rescale_to_next_inplace(*context, ct);
        }
    }

    void PhantomCrypto::_square(PhantomCiphertext &result, PhantomCiphertext const &ct) {
        result = phantom::multiply_and_relin(*context, ct, ct, relinKey);
        if (this->params->schemeType == HEScheme::CKKS) {
            phantom::rescale_to_next_inplace(*context, result);
        }
    }

    void PhantomCrypto::_power(PhantomCiphertext &result, PhantomCiphertext const &ct, int const &n) {
        throw std::logic_error("Not implemented");
    }

    void PhantomCrypto::_rotate(PhantomCiphertext &result, PhantomCiphertext const &ct, int const &step) {
        result = phantom::rotate(*context, ct, step, galoisKeys);
    }

    void PhantomCrypto::_rotate_columns(PhantomCiphertext &result, PhantomCiphertext const &ct) {
        throw std::logic_error("Not implemented");
    }

    void PhantomCrypto::_shift(PhantomCiphertext &result, PhantomCiphertext const &ct, int const &n) {
        throw std::logic_error("Not implemented");
    }

    void PhantomCrypto::_and(PhantomCiphertext &result, PhantomCiphertext const &ct_1, PhantomCiphertext const &ct_2) {
        _multiply(result, ct_1, ct_2);
    }

    void PhantomCrypto::_or(PhantomCiphertext &result, PhantomCiphertext const &ct_1, PhantomCiphertext const &ct_2) {
        PhantomCiphertext mult12;
        _multiply(mult12, ct_1, ct_2);
        PhantomCiphertext add12;
        _add(add12, ct_1, ct_2);
        parms_unify(add12, mult12);
        _sub(result, mult12, add12);
    }

    void PhantomCrypto::_xor(PhantomCiphertext &result, PhantomCiphertext const &ct_1, PhantomCiphertext const &ct_2) {
        PhantomCiphertext mult12;
        _multiply(mult12, ct_1, ct_2);
        PhantomPlaintext ptxt;
        _encode(ptxt, 2.0);
        parms_unify(ptxt, mult12);
        _multiply_plain(mult12, ptxt);
        PhantomCiphertext add12;
        _add(add12, ct_1, ct_2);
        parms_unify(add12, mult12);
        _sub(result, mult12, add12);
    }

    void PhantomCrypto::_xnor(PhantomCiphertext &result, PhantomCiphertext const &ct_1, PhantomCiphertext const &ct_2) {
        _not(result, ct_2);
        _xor(result, ct_1, result);
    }

    void PhantomCrypto::_not(PhantomCiphertext &result, PhantomCiphertext const &ct) {
        PhantomPlaintext ptxt;
        _encode(ptxt, -1.0);
        PhantomCiphertext one;
        _encrypt(one, ptxt);
        _sub(result, one, ct);
    }

    void PhantomCrypto::_mask(PhantomCiphertext &result, PhantomCiphertext const &ct, int const &index) {
        throw std::logic_error("Not implemented");
    }

    void PhantomCrypto::_mask(PhantomCiphertext &result, PhantomCiphertext const &ct, std::vector<int> const &indices) {
        throw std::logic_error("Not implemented");
    }

    void PhantomCrypto::_total_sum(PhantomCiphertext &result, PhantomCiphertext const &ct) {
        throw std::logic_error("Not implemented");
    }

    void PhantomCrypto::_running_sum(PhantomCiphertext &result, PhantomCiphertext const &ct) {
        throw std::logic_error("Not implemented");
    }

    PhantomCrypto::PhantomCrypto() {
        this->publicKey = PhantomPublicKey();
        this->secretKey = PhantomSecretKey();
        this->relinKey = PhantomRelinKey();
        this->galoisKeys = PhantomGaloisKey();

        // Pointers
        this->phantomParams = nullptr;
        this->context = nullptr;
        this->ckksEncoder = nullptr;
        this->batchEncoder = nullptr;
    }
    PhantomCrypto::PhantomCrypto(CryptoParams &params) : PhantomCrypto() {
        this->update_encryption_params(params);
        this->initPhantomParams();
        this->context = std::make_shared<PhantomContext>(*this->phantomParams);
        this->set_encoder(this->params->schemeType);
    }

    PhantomCrypto::PhantomCrypto(const std::string &param_string) : PhantomCrypto() {
        CryptoParams params;
        rapidjson::Document doc;
        parse_params(params, doc, param_string);
        this->update_encryption_params(params);
        this->initPhantomParams();
        this->context = std::make_shared<PhantomContext>(*this->phantomParams);
        this->set_encoder(this->params->schemeType);
    }

    PhantomCrypto::~PhantomCrypto() = default;

    void PhantomCrypto::GenerateKeyPair(const std::string &pubKeyFilename, const std::string &secKeyFilename) {
        this->secretKey = PhantomSecretKey(*this->context);
        this->publicKey = this->secretKey.gen_publickey(*this->context);
        this->relinKey = this->secretKey.gen_relinkey(*this->context);
        this->galoisKeys = this->secretKey.create_galois_keys(*this->context);

        // save keys to file
        const std::shared_ptr<FSManager> secretKeyFS = FSManager::createFSManager(secKeyFilename);
        const std::shared_ptr<FSManager> publicKeyFS = FSManager::createFSManager(pubKeyFilename);
        publicKeyFS->OpenOutputStream();
        secretKeyFS->OpenOutputStream();
        this->publicKey.save(publicKeyFS->GetOutputStream());
        this->secretKey.save(secretKeyFS->GetOutputStream());
        publicKeyFS->CloseOutputStream();
        secretKeyFS->CloseOutputStream();

        const std::shared_ptr<FSManager> relinKeyFS = FSManager::createFSManager(pubKeyFilename + ".relin");
        relinKeyFS->OpenOutputStream();
        this->relinKey.save(relinKeyFS->GetOutputStream());
        relinKeyFS->CloseOutputStream();
        const std::shared_ptr<FSManager> galoisKeyFS = FSManager::createFSManager(pubKeyFilename + ".galois");
        galoisKeyFS->OpenOutputStream();
        this->galoisKeys.save(galoisKeyFS->GetOutputStream());
        galoisKeyFS->CloseOutputStream();
    }

    void PhantomCrypto::LoadPublicKey(const std::string &pubKeyFilename) {
        const std::shared_ptr<FSManager> publicKeyFS = FSManager::createFSManager(pubKeyFilename);
        publicKeyFS->OpenInputStream();
        this->publicKey.load(publicKeyFS->GetInputStream());
        publicKeyFS->CloseInputStream();
        const std::shared_ptr<FSManager> relinKeyFS = FSManager::createFSManager(pubKeyFilename + ".relin");
        relinKeyFS->OpenInputStream();
        this->relinKey.load(relinKeyFS->GetInputStream());
        relinKeyFS->CloseInputStream();
        const std::shared_ptr<FSManager> galoisKeyFS = FSManager::createFSManager(pubKeyFilename + ".galois");
        galoisKeyFS->OpenInputStream();
        this->galoisKeys.load(galoisKeyFS->GetInputStream());
        galoisKeyFS->CloseInputStream();
    }

    PlainText PhantomCrypto::Encode(double d) {
        PhantomPlaintext ptxt = PhantomPlaintext();
        this->_encode(ptxt, d);
        return PlainText(ptxt);
    }

    PlainText PhantomCrypto::Encode(long l) {
        PhantomPlaintext ptxt = PhantomPlaintext();
        this->_encode(ptxt, l);
        return PlainText(ptxt);
    }

    PlainText PhantomCrypto::Encode(const std::string &str) {
        if (this->params->schemeType == HECrypto::HEScheme::BFV) {
            return this->Encode(stol(str));
        } else if (this->params->schemeType == HECrypto::HEScheme::CKKS) {
            return this->Encode(stod(str));
        } else {
            throw invalid_argument("Invalid scheme type");
        }
    }
    PlainText PhantomCrypto::Encode(const std::vector<double> &vec) {
        PhantomPlaintext ptxt = PhantomPlaintext();
        this->ckksEncoder->encode(*this->context, vec, this->params->scale, ptxt);
        return PlainText(ptxt);
    }
    PlainText PhantomCrypto::Encode(const std::vector<long> &vec) {
        PhantomPlaintext ptxt = PhantomPlaintext();
        // transform long to uint_64
        vector<ulong> uvec;
        for (auto &l : vec) {
            uvec.push_back(static_cast<ulong>(l));
        }
        this->batchEncoder->encode(*this->context, uvec, ptxt);
        return PlainText(ptxt);
    }

    std::vector<PlainText> PhantomCrypto::EncodeMany(const std::vector<double> &vec) {
        vector<PlainText> ptxt_vec;
        if (this->slot_count >= vec.size()) {
            ptxt_vec.emplace_back(this->Encode(vec));
        } else {
            this->createPlainVector<double>(ptxt_vec, vec);
        }
        return ptxt_vec;
    }
    std::vector<PlainText> PhantomCrypto::EncodeMany(const std::vector<long> &vec) {
        vector<PlainText> ptxt_vec;
        if (this->slot_count >= vec.size()) {
            ptxt_vec.emplace_back(this->Encode(vec));
        } else {
            this->createPlainVector<long>(ptxt_vec, vec);
        }
        return ptxt_vec;
    }

    CipherText PhantomCrypto::Encrypt(const PlainText &pt) {
        PhantomPlaintext ptxt = PhantomPlaintext();
        PhantomCiphertext ctxt = PhantomCiphertext();
        this->toPhantomPlaintext(ptxt, pt);
        this->_encrypt(ctxt, ptxt);
        return CipherText(ctxt);
    }

    std::string PhantomCrypto::Encrypt(const std::string &str) {
        return this->Encrypt(PlainText(str)).toString();
    }

    std::string PhantomCrypto::Encrypt(const std::vector<std::string> &spt_vec) {
        PlainText pt;
        if (this->params->schemeType == HECrypto::HEScheme::BFV) {
            vector<long> vec;
            vec.resize(spt_vec.size());
            transform(spt_vec.begin(), spt_vec.end(), vec.begin(), [](string const &s) { return stol(s); });
            pt = this->Encode(vec);
        } else if (this->params->schemeType == HECrypto::HEScheme::CKKS) {
            vector<double> vec;
            vec.resize(spt_vec.size());
            transform(spt_vec.begin(), spt_vec.end(), vec.begin(), [](string const &s) { return stod(s); });
            pt = this->Encode(vec);
        }
        return this->Encrypt(pt).toString();
    }

    void PhantomCrypto::LoadSecretKey(const std::string &secKeyFilename) {
        const std::shared_ptr<FSManager> secretKeyFS = FSManager::createFSManager(secKeyFilename);
        secretKeyFS->OpenInputStream();
        this->secretKey.load(secretKeyFS->GetInputStream());
        secretKeyFS->CloseInputStream();
    }

    void PhantomCrypto::Decode(std::vector<double> &vec, const PlainText &pt) {
        PhantomPlaintext ptxt;
        this->toPhantomPlaintext(ptxt, pt);
        this->ckksEncoder->decode(*this->context, ptxt, vec);
    }

    void PhantomCrypto::Decode(std::vector<long> &vec, const PlainText &pt) {
        PhantomPlaintext ptxt;
        this->toPhantomPlaintext(ptxt, pt);
        vector<ulong>  ul_vector = this->batchEncoder->decode(*this->context, ptxt);
        for (auto &ul : ul_vector) {
            vec.push_back(static_cast<long>(ul));
        }
    }

    PlainText PhantomCrypto::Decrypt(const CipherText &ct, bool noBatching) {
        PhantomCiphertext ctxt;
        PhantomPlaintext ptxt;
        this->toPhantomCiphertext(ctxt, ct);
        this->_decrypt(ptxt, ctxt);
        return PlainText(ptxt);
    }

    std::string PhantomCrypto::Decrypt(const std::string &sct, bool noBatching) {
        return this->Decrypt(CipherText(sct), noBatching).toString();
    }

    CipherText PhantomCrypto::toCipherText(const std::string &str) {
        PhantomCiphertext ctxt;
        base64::decoder decoder;
        stringstream decoded, in(str);
        decoder.decode(in, decoded);
        ctxt.load(decoded);
        return CipherText(ctxt);
    }

    std::vector<CipherText> PhantomCrypto::toCipherText(const std::vector<std::string> &strs) {
        vector<CipherText> vec_ct;
        transform(
            strs.begin(), strs.end(), back_inserter(vec_ct), [this](string const &s) { return this->toCipherText(s); });
        return vec_ct;
    }

    PlainText PhantomCrypto::toPlainText(const std::string &str) {
        PhantomPlaintext ptxt;
        stringstream ss(str);
        ptxt.load(ss);
        return PlainText(ptxt);
    }

    std::vector<PlainText> PhantomCrypto::toPlainText(const std::vector<std::string> &strs) {
        vector<PlainText> vec_pt;
        transform(
            strs.begin(), strs.end(), back_inserter(vec_pt), [this](string const &s) { return this->toPlainText(s); });
        return vec_pt;
    }

    CipherText PhantomCrypto::Add(const CipherText &ct_1, const CipherText &ct_2) {
        PhantomCiphertext ctxt_1, ctxt_2;
        this->toPhantomCiphertext(ctxt_1, ct_1);
        this->toPhantomCiphertext(ctxt_2, ct_2);

        this->_add(ctxt_1, ctxt_2);
        return CipherText(ctxt_1);
    }

    std::string PhantomCrypto::Add(const std::string &sct_1, const std::string &sct_2) {
        CipherText ct_1 = CipherText(sct_1);
        CipherText ct_2 = CipherText(sct_2);
        CipherText result = this->Add(ct_1, ct_2);
        return result.toString();
    }

    CipherText PhantomCrypto::AddPlain(const CipherText &ct, const PlainText &pt) {
        PhantomCiphertext ctxt;
        PhantomPlaintext ptxt;
        this->toPhantomCiphertext(ctxt, ct);
        this->toPhantomPlaintext(ptxt, pt);
        this->_add_plain(ctxt, ptxt);
        return CipherText(ctxt);
    }

    std::string PhantomCrypto::AddPlain(const std::string &sct, const std::string &spt) {
        PlainText pt = PlainText(spt);
        CipherText ct = CipherText(sct);
        CipherText result = this->AddPlain(ct, pt);
        return result.toString();
    }

    std::vector<CipherText> PhantomCrypto::FullAdder(
        const std::vector<CipherText> &vec_ct_1,
        const std::vector<CipherText> &vec_ct_2,
        size_t max_count) {
        vector<PhantomCiphertext> vec_ctxt_1;
        vector<PhantomCiphertext> vec_ctxt_2;
        vector<PhantomCiphertext> vec_ctxt_result;
        this->toPhantomCiphertext(vec_ctxt_1, vec_ct_1);
        this->toPhantomCiphertext(vec_ctxt_2, vec_ct_2);
        this->_full_adder(vec_ctxt_result, vec_ctxt_1, vec_ctxt_2, max_count);

        vector<CipherText> vec_result;
        toCipherText(vec_result, vec_ctxt_result);
        return vec_result;
    }

    std::vector<std::string> PhantomCrypto::FullAdder(
        const std::vector<std::string> &vec_sct_1,
        const std::vector<std::string> &vec_sct_2,
        size_t max_count) {
        vector<CipherText> ct_1 = this->toCipherText(vec_sct_1);
        vector<CipherText> ct_2 = this->toCipherText(vec_sct_2);
        vector<CipherText> result = this->FullAdder(ct_1, ct_2, max_count);
        vector<string> sresult;
        transform(result.begin(), result.end(), back_inserter(sresult), [](CipherText &ct) { return ct.toString(); });
        return sresult;
    }

    CipherText PhantomCrypto::Multiply(const CipherText &ct_1, const CipherText &ct_2) {
        PhantomCiphertext ctxt_1, ctxt_2;
        this->toPhantomCiphertext(ctxt_1, ct_1);
        this->toPhantomCiphertext(ctxt_2, ct_2);

        this->_multiply(ctxt_1, ctxt_2);
        return CipherText(ctxt_1);
    }

    std::string PhantomCrypto::Multiply(const std::string &sct_1, const std::string &sct_2) {
        CipherText ct_1 = CipherText(sct_1);
        CipherText ct_2 = CipherText(sct_2);
        CipherText result = this->Multiply(ct_1, ct_2);
        return result.toString();
    }

    CipherText PhantomCrypto::MultiplyPlain(const CipherText &ct, const PlainText &pt) {
        PhantomCiphertext ctxt;
        PhantomPlaintext ptxt;
        this->toPhantomCiphertext(ctxt, ct);
        this->toPhantomPlaintext(ptxt, pt);
        this->_multiply_plain(ctxt, ptxt);
        return CipherText(ctxt);
    }

    std::string PhantomCrypto::MultiplyPlain(const std::string &sct, const std::string &spt) {
        PlainText pt = PlainText(spt);
        CipherText ct = CipherText(sct);
        CipherText result = this->MultiplyPlain(ct, pt);
        return result.toString();
    }

    CipherText PhantomCrypto::Square(const CipherText &ct) {
        PhantomCiphertext ctxt, result;
        this->toPhantomCiphertext(ctxt, ct);
        this->_square(result, ctxt);
        return CipherText(result);
    }

    std::string PhantomCrypto::Square(const std::string &sct) {
        CipherText ct = CipherText(sct);
        CipherText result = this->Square(ct);
        return result.toString();
    }

    CipherText PhantomCrypto::Power(const CipherText &ct, const int &n) {
        if (this->params->schemeType == HECrypto::HEScheme::CKKS) {
            throw invalid_argument("Power operation is not supported in CKKS scheme");
        }
        PhantomCiphertext ctxt = PhantomCiphertext();
        PhantomCiphertext result = PhantomCiphertext();
        this->toPhantomCiphertext(ctxt, ct);
        this->_power(result, ctxt, n);
        return CipherText(result);
    }

    std::string PhantomCrypto::Power(const std::string &sct, const int &n) {
        CipherText ct = CipherText(sct);
        CipherText result = this->Power(ct, n);
        return result.toString();
    }

    CipherText PhantomCrypto::Subtract(const CipherText &ct_1, const CipherText &ct_2) {
        PhantomCiphertext ctxt_1, ctxt_2;
        this->toPhantomCiphertext(ctxt_1, ct_1);
        this->toPhantomCiphertext(ctxt_2, ct_2);
        this->_sub(ctxt_1, ctxt_2);
        return CipherText(ctxt_1);
    }

    std::string PhantomCrypto::Subtract(const std::string &sct_1, const std::string &sct_2) {
        CipherText ct_1 = CipherText(sct_1);
        CipherText ct_2 = CipherText(sct_2);
        CipherText result = this->Subtract(ct_1, ct_2);
        return result.toString();
    }

    CipherText PhantomCrypto::Rotate(const CipherText &ct, const int &n) {
        PhantomCiphertext ctxt, result;
        this->toPhantomCiphertext(ctxt, ct);
        this->_rotate(result, ctxt, n);
        return CipherText(result);
    }

    std::string PhantomCrypto::Rotate(const std::string &sct, const int &n) {
        CipherText ct = CipherText(sct);
        CipherText result = this->Rotate(ct, n);
        return result.toString();
    }

    CipherText PhantomCrypto::Shift(const CipherText &ct, const int &n) {
        if (this->params->schemeType == HECrypto::HEScheme::CKKS) {
            throw invalid_argument("Power operation is not supported in CKKS scheme");
        }

        PhantomCiphertext ctxt = PhantomCiphertext();
        PhantomCiphertext result = PhantomCiphertext();
        this->toPhantomCiphertext(ctxt, ct);
        this->_shift(result, ctxt, n);
        return CipherText(result);
    }

    std::string PhantomCrypto::Shift(const std::string &sct, const int &n) {
        CipherText ct = CipherText(sct);
        CipherText result = this->Shift(ct, n);
        return result.toString();
    }

    CipherText PhantomCrypto::RotateColumns(const CipherText &ct) {
        PhantomCiphertext ctxt, result;
        this->toPhantomCiphertext(ctxt, ct);
        this->_rotate_columns(result, ctxt);
        return CipherText(result);
    }

    std::string PhantomCrypto::RotateColumns(const std::string &sct) {
        CipherText ct = CipherText(sct);
        CipherText result = this->RotateColumns(ct);
        return result.toString();
    }

    CipherText PhantomCrypto::And(const CipherText &ct_1, const CipherText &ct_2) {
        PhantomCiphertext ctxt_1, ctxt_2, result;
        this->toPhantomCiphertext(ctxt_1, ct_1);
        this->toPhantomCiphertext(ctxt_2, ct_2);

        this->_and(result, ctxt_1, ctxt_2);
        return CipherText(ctxt_1);
    }

    std::string PhantomCrypto::And(const std::string &sct_1, const std::string &sct_2) {
        CipherText ct_1 = CipherText(sct_1);
        CipherText ct_2 = CipherText(sct_2);
        CipherText result = this->And(ct_1, ct_2);
        return result.toString();
    }

    CipherText PhantomCrypto::Or(const CipherText &ct_1, const CipherText &ct_2) {
        PhantomCiphertext ctxt_1, ctxt_2, result;
        this->toPhantomCiphertext(ctxt_1, ct_1);
        this->toPhantomCiphertext(ctxt_2, ct_2);

        this->_or(result, ctxt_1, ctxt_2);
        return CipherText(ctxt_1);
    }

    std::string PhantomCrypto::Or(const std::string &sct_1, const std::string &sct_2) {
        CipherText ct_1 = CipherText(sct_1);
        CipherText ct_2 = CipherText(sct_2);
        CipherText result = this->Or(ct_1, ct_2);
        return result.toString();
    }

    CipherText PhantomCrypto::Xor(const CipherText &ct_1, const CipherText &ct_2) {
        PhantomCiphertext ctxt_1, ctxt_2, result;
        this->toPhantomCiphertext(ctxt_1, ct_1);
        this->toPhantomCiphertext(ctxt_2, ct_2);

        this->_xor(result, ctxt_1, ctxt_2);
        return CipherText(ctxt_1);
    }

    std::string PhantomCrypto::Xor(const std::string &sct_1, const std::string &sct_2) {
        CipherText ct_1 = CipherText(sct_1);
        CipherText ct_2 = CipherText(sct_2);
        CipherText result = this->Xor(ct_1, ct_2);
        return result.toString();
    }

    CipherText PhantomCrypto::Xnor(const CipherText &ct_1, const CipherText &ct_2) {
        PhantomCiphertext ctxt_1, ctxt_2, result;
        this->toPhantomCiphertext(ctxt_1, ct_1);
        this->toPhantomCiphertext(ctxt_2, ct_2);

        this->_xnor(result, ctxt_1, ctxt_2);
        return CipherText(ctxt_1);
    }

    std::string PhantomCrypto::Xnor(const std::string &sct_1, const std::string &sct_2) {
        CipherText ct_1 = CipherText(sct_1);
        CipherText ct_2 = CipherText(sct_2);
        CipherText result = this->Xnor(ct_1, ct_2);
        return result.toString();
    }

    CipherText PhantomCrypto::Not(const CipherText &ct) {
        PhantomCiphertext ctxt, result;
        this->toPhantomCiphertext(ctxt, ct);
        this->_not(result, ctxt);
        return CipherText(result);
    }

    std::string PhantomCrypto::Not(const std::string &sct) {
        CipherText ct = CipherText(sct);
        CipherText result = this->Not(ct);
        return result.toString();
    }

    CipherText PhantomCrypto::Mask(const CipherText &ct, const int &index) {
        PhantomCiphertext ctxt, result;
        this->toPhantomCiphertext(ctxt, ct);
        this->_mask(result, ctxt, index);
        return CipherText(result);
    }

    CipherText PhantomCrypto::Mask(const CipherText &ct, const std::vector<int> &indices) {
        PhantomCiphertext ctxt, result;
        this->toPhantomCiphertext(ctxt, ct);
        this->_mask(result, ctxt, indices);
        return CipherText(result);
    }

    std::string PhantomCrypto::Mask(const std::string &sct, const int &index) {
        CipherText ct = CipherText(sct);
        CipherText result = this->Mask(ct, index);
        return result.toString();
    }

    std::string PhantomCrypto::Mask(const std::string &sct, const std::vector<int> &indices) {
        CipherText ct = CipherText(sct);
        CipherText result = this->Mask(ct, indices);
        return result.toString();
    }

    CipherText PhantomCrypto::IsEqual(const CipherText &ct_1, const CipherText &ct_2) {
        PhantomCiphertext ctxt_1, ctxt_2, result;
        this->toPhantomCiphertext(ctxt_1, ct_1);
        this->toPhantomCiphertext(ctxt_2, ct_2);
        this->_xnor(result, ctxt_1, ctxt_2);
        return CipherText(result);
    }

    std::string PhantomCrypto::IsEqual(const std::string &sct_1, const std::string &sct_2) {
        CipherText ct_1 = CipherText(sct_1);
        CipherText ct_2 = CipherText(sct_2);
        CipherText result = this->IsEqual(ct_1, ct_2);
        return result.toString();
    }

    CipherText PhantomCrypto::RunningSum(const CipherText &ct) {
        PhantomCiphertext ctxt, result;
        this->toPhantomCiphertext(ctxt, ct);
        this->_running_sum(result, ctxt);
        return CipherText(result);
    }

    std::string PhantomCrypto::RunningSum(const std::string &sct) {
        CipherText ct = CipherText(sct);
        CipherText result = this->RunningSum(ct);
        return result.toString();
    }

    CipherText PhantomCrypto::TotalSum(const CipherText &ct) {
        PhantomCiphertext ctxt, result;
        this->toPhantomCiphertext(ctxt, ct);
        this->_total_sum(result, ctxt);
        return CipherText(result);
    }

    std::string PhantomCrypto::TotalSum(const std::string &sct) {
        CipherText ct = CipherText(sct);
        CipherText result = this->TotalSum(ct);
        return result.toString();
    }

    CipherText PhantomCrypto::buildCipherText(const std::string &str) const {
        PhantomCiphertext ctxt;
        base64::decoder decoder;
        stringstream decoded, in(str);
        decoder.decode(in, decoded);
        ctxt.load(decoded);
        return CipherText(ctxt);
    }

    template <typename T>
    void PhantomCrypto::createPlainVector(std::vector<PlainText> &vec, std::vector<T> const &data) {
        int splits = (data.size() - 1) / this->slot_count + 1;
        auto start = data.begin();
        for (int i = 0; i < splits; i++) {
            auto end = start + this->slot_count;
            if (end > data.end()) {
                end = data.end();
            }
            vector<T> sub_data(start, end);
            vec.emplace_back(this->Encode(sub_data));
            start += this->slot_count;
        }
    }
    void PhantomCrypto::toPhantomCiphertext(PhantomCiphertext &ct, CipherText const &c) {
        CipherTextData data = c.getData();
        if (holds_alternative<string>(data)) {
            string s = get<string>(data);
            base64::decoder decoder;
            stringstream decoded, in(s);
            decoder.decode(in, decoded);
            ct.load(decoded);
        } else if (holds_alternative<PhantomCiphertext>(data)) {
            ct = get<PhantomCiphertext>(data);
        } else {
            throw invalid_argument("Invalid CipherTextData type");
        }
    }

    void PhantomCrypto::toPhantomCiphertext(
        std::vector<PhantomCiphertext> &vec_ct,
        std::vector<CipherText> const &vec_c) {
        transform(vec_c.begin(), vec_c.end(), back_inserter(vec_ct), [this](CipherText const &c) {
            PhantomCiphertext ct;
            this->toPhantomCiphertext(ct, c);
            return ct;
        });
    }
    void PhantomCrypto::toPhantomPlaintext(PhantomPlaintext &pt, PlainText const &p) {
        PlainTextData data = p.getData();

        if (holds_alternative<int>(data)) {
            int value = get<int>(data);
            vector<ulong> vec = {static_cast<ulong>(value)};
            this->batchEncoder->encode(*context, vec, pt);
        } else if (holds_alternative<string>(data)) {
            string s = get<string>(data);
            if (this->params->schemeType == HECrypto::HEScheme::BFV) {
                istringstream iss(s);
                vector<ulong> vec((istream_iterator<ulong>(iss)), istream_iterator<ulong>());
                this->batchEncoder->encode(*context, vec, pt);
            }
        } else if (holds_alternative<PhantomPlaintext>(data)) {
            pt = get<PhantomPlaintext>(data);
        } else {
            throw invalid_argument("Invalid PlainTextData type");
        }
    }
    void PhantomCrypto::toPhantomPlaintext(std::vector<PhantomPlaintext> &vec_pt, std::vector<PlainText> const &vec_p) {
        transform(vec_p.begin(), vec_p.end(), back_inserter(vec_pt), [this](PlainText const &p) {
            PhantomPlaintext pt;
            this->toPhantomPlaintext(pt, p);
            return pt;
        });
    }
    void PhantomCrypto::toCipherText(CipherText &c, PhantomCiphertext const &ct) const {
        c.setData(ct);
    }
    void PhantomCrypto::toCipherText(std::vector<CipherText> &vec_c, std::vector<PhantomCiphertext> const &vec_ct)
        const {
        transform(vec_ct.begin(), vec_ct.end(), back_inserter(vec_c), [this](PhantomCiphertext const &ct) {
            CipherText c;
            toCipherText(c, ct);
            return c;
        });
    }

    void PhantomCrypto::toPlainText(PlainText &p, PhantomPlaintext const &pt) {
        throw std::logic_error("Not implemented");
    }
    void PhantomCrypto::toPlainText(std::vector<PlainText> &vec_p, std::vector<PhantomPlaintext> const &vec_pt) {
        throw std::logic_error("Not implemented");
    }
}  // namespace SpatialFHE