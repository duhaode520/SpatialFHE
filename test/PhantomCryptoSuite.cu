//
// Created by ubuntu on 11/26/24.
//
#include <gtest/gtest.h>
#include <phantom.h>
#include <vector>
#include "CipherText.h"
#include "HECrypto.h"

#define private public
#include "PhantomCrypto.cuh"
#undef private

using namespace SpatialFHE;

class PhantomCryptoCKKSSuite : public testing::Test {
protected:
    static PhantomCrypto* crypto;
    static std::string json;
    constexpr static const double ERROR = 1e-4;  // scale factor is 30, so the error is 1e-4 ~ 2^-10
    static std::string PUBKEY_PATH;
    static std::string SECKEY_PATH;
    static void SetUpTestSuite() {
        hipSetDevice(2);
        json = R"(
            {
                    "CoeffModulusBits": [60, 40, 40, 40, 40, 40, 40, 40, 60],
                    "SchemeType": "CKKS",
                    "PolyModulusDegree": 16384,
                    "PlaintextModulus": 0,
                    "CoeffModulusPrimes": [0],
                    "ScaleFactor": 40
            })";

        crypto = new PhantomCrypto(json);
        PUBKEY_PATH = "public.key";
        SECKEY_PATH = "secret.key";
        crypto->GenerateKeyPair(PUBKEY_PATH, SECKEY_PATH);
    }

    static void TearDownTestSuite() {
        delete crypto;
    }
};

// CUDA 对象不能成为全局静态变量，析构时会有 hipErrorDeinitialized 的问题，要手动控制生命周期
PhantomCrypto* PhantomCryptoCKKSSuite::crypto;
std::string PhantomCryptoCKKSSuite::json, PhantomCryptoCKKSSuite::PUBKEY_PATH, PhantomCryptoCKKSSuite::SECKEY_PATH;

TEST_F(PhantomCryptoCKKSSuite, EncodeDecode) {
    PlainText pt = crypto->Encode(1.0);
    std::vector<double> result;
    crypto->Decode(result, pt);
    ASSERT_EQ(1.0, result[0]);
}

TEST_F(PhantomCryptoCKKSSuite, BatchEncodeDecode) {
    std::vector<double> vec = {1.0, 2.0, 3.0};
    std::vector<PlainText> pts = crypto->EncodeMany(vec);
    std::vector<double> result;
    if (pts.size() == 1) {
        crypto->Decode(result, pts[0]);
        ASSERT_NEAR(1.0, result[0], ERROR);
        ASSERT_NEAR(2.0, result[1], ERROR);
        ASSERT_NEAR(3.0, result[2], ERROR);
    }
}

TEST_F(PhantomCryptoCKKSSuite, BigBatchEncodeDecode) {
    std::vector<double> vec(8192, 1.0);
    for (int i = 0; i < 8192; i++) {
        vec[i] = i;
    }
    std::vector<PlainText> pts = crypto->EncodeMany(vec);
    ASSERT_EQ(pts.size(), 1);
    for (int i = 0; i < pts.size(); i++) {
        std::vector<double> result;
        crypto->Decode(result, pts[i]);
        for (int j = 0; j < 8192; j++) {
            ASSERT_NEAR(result[j], vec[j + i * 8192], ERROR);
        }
    }
}

TEST_F(PhantomCryptoCKKSSuite, EncryptDecrypt) {
    PlainText pt = crypto->Encode(1.0);
    CipherText ct = crypto->Encrypt(pt);
    PlainText res_pt = crypto->Decrypt(ct);
    std::vector<double> result;
    crypto->Decode(result, res_pt);
    ASSERT_NEAR(1.0, result[0], ERROR);
}

TEST_F(PhantomCryptoCKKSSuite, Add) {
    PlainText pt1 = crypto->Encode(1.0);
    PlainText pt2 = crypto->Encode(2.0);
    CipherText ct1 = crypto->Encrypt(pt1);
    CipherText ct2 = crypto->Encrypt(pt2);
    CipherText ct3 = crypto->Add(ct1, ct2);
    PlainText res_pt = crypto->Decrypt(ct3);
    std::vector<double> result;
    crypto->Decode(result, res_pt);
    ASSERT_NEAR(3.0, result[0], ERROR);
}

TEST_F(PhantomCryptoCKKSSuite, ReloadAndAdd) {
    PhantomCrypto crypto2 = PhantomCrypto(json);
    crypto2.LoadKeyPair(PUBKEY_PATH, SECKEY_PATH);
    PlainText pt1 = crypto2.Encode(1.0);
    PlainText pt2 = crypto2.Encode(2.0);
    CipherText ct1 = crypto2.Encrypt(pt1);
    CipherText ct2 = crypto2.Encrypt(pt2);
    CipherText ct3 = crypto2.Add(ct1, ct2);
    PlainText res_pt = crypto2.Decrypt(ct3);
    std::vector<double> result;
    crypto2.Decode(result, res_pt);
    ASSERT_NEAR(3.0, result[0], ERROR);
}

TEST_F(PhantomCryptoCKKSSuite, AddInside) {
    PlainText pt1 = crypto->Encode(1.0);
    PlainText pt2 = crypto->Encode(2.0);
    CipherText ct1 = crypto->Encrypt(pt1);
    CipherText ct2 = crypto->Encrypt(pt2);
    PhantomCiphertext ct_1 = std::get<PhantomCiphertext>(ct1.getData());
    PhantomCiphertext ct_2 = std::get<PhantomCiphertext>(ct2.getData());
    PhantomCiphertext ct = PhantomCiphertext();
    crypto->_add(ct, ct_1, ct_2);
    std::cout << "ct3 scale: " << ct.scale() << std::endl;
    CipherText ct3 = CipherText(ct);

    PlainText res_pt = crypto->Decrypt(ct3);
    std::vector<double> result;
    crypto->Decode(result, res_pt);
    ASSERT_NEAR(3.0, result[0], ERROR);
}

TEST_F(PhantomCryptoCKKSSuite, AddPlain) {
    PlainText pt1 = crypto->Encode(1.0);
    PlainText pt2 = crypto->Encode(2.0);
    CipherText ct1 = crypto->Encrypt(pt1);
    CipherText ct3 = crypto->AddPlain(ct1, pt2);
    PlainText res_pt = crypto->Decrypt(ct3);
    std::vector<double> result;
    crypto->Decode(result, res_pt);
    ASSERT_NEAR(3.0, result[0], ERROR);
}

TEST_F(PhantomCryptoCKKSSuite, Sub) {
    PlainText pt1 = crypto->Encode(1.0);
    PlainText pt2 = crypto->Encode(2.0);
    CipherText ct1 = crypto->Encrypt(pt1);
    CipherText ct2 = crypto->Encrypt(pt2);
    CipherText ct3 = crypto->Subtract(ct1, ct2);
    PlainText res_pt = crypto->Decrypt(ct3);
    std::vector<double> result;
    crypto->Decode(result, res_pt);
    ASSERT_NEAR(-1.0, result[0], ERROR);
}

TEST_F(PhantomCryptoCKKSSuite, Multiply) {
    PlainText pt1 = crypto->Encode(1.0);
    PlainText pt2 = crypto->Encode(2.0);
    CipherText ct1 = crypto->Encrypt(pt1);
    CipherText ct2 = crypto->Encrypt(pt2);
    CipherText ct3 = crypto->Multiply(ct1, ct2);
    PlainText res_pt = crypto->Decrypt(ct3);
    std::vector<double> result;
    crypto->Decode(result, res_pt);
    ASSERT_NEAR(2.0, result[0], ERROR);
}

TEST_F(PhantomCryptoCKKSSuite, MultiplyPlain) {
    PlainText pt1 = crypto->Encode(1.0);
    PlainText pt2 = crypto->Encode(2.0);
    CipherText ct1 = crypto->Encrypt(pt1);
    CipherText ct3 = crypto->MultiplyPlain(ct1, pt2);
    PlainText res_pt = crypto->Decrypt(ct3);
    std::vector<double> result;
    crypto->Decode(result, res_pt);
    ASSERT_NEAR(2.0, result[0], ERROR);
}
